// Compile :
// nvcc -o bruteforce bruteforce.cu

// Launch (within screen, writing to file as well for safety):
// screen
// ./bruteforce | tee resultats.txt
// Ctrl+A and d to detach from the screen session

// To reattach to the screen session:
// screen -ls
// screen -r <session_id>

#include <stdio.h>
#include <stdint.h>
#include <time.h>
#include <hip/hip_runtime.h>


// Macro to check CUDA errors
#define CUDA_CHECK(err) \
if (err != hipSuccess) { \
        fprintf(stderr, "[X] CUDA Error: %s (code %d), line %d\n", hipGetErrorString(err), err, __LINE__); \
        exit(EXIT_FAILURE); \
    }

// Charset
__constant__ char d_alphabet[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789_-"; // prod
const int alphabet_len = 2 * 26 + 10 + 2; // !! Make sure to update this if you change the alphabet !!
// __constant__ char d_alphabet[] = "pasword"; // testing
// const int alphabet_len = 7; // !! Make sure to update this if you change the alphabet !!

// WASM data
uint8_t wasm_data_at_8[] = {0xDC, 0x87, 0xDB, 0x6B, 0x7C, 0xFD, 0x6D, 0x20};
uint8_t wasm_data_at_16[] = {0x8B, 0xC9, 0xDA, 0x58, 0xF2, 0xBF, 0x1E, 0xA1}; // prod
// uint8_t wasm_data_at_16[] = {0x2C, 0xE4, 0x0E, 0x7B, 0x77, 0x02, 0x1A, 0x5D}; // testing

// check function, for GPU
__device__ int check_gpu(const char *input_text, const uint8_t *wasm_data_at_8, const uint8_t *wasm_data_at_16) {
    uint8_t memory[24] = {0};

    for (int i = 0; i < 8; i++) memory[i] = input_text[i];
    for (int i = 0; i < 8; i++) memory[8 + i] = wasm_data_at_8[i];
    for (int i = 0; i < 8; i++) memory[16 + i] = wasm_data_at_16[i];

    for (int var1 = 0; var1 < 8; var1++) {
        uint8_t var5 = memory[var1] ^ memory[8 + var1];

        uint32_t temp_loaded_i32 = 0;
        for (int b = 0; b < 4; b++) {
            temp_loaded_i32 |= ((uint32_t)memory[var1 + b]) << (8 * b);
        }

        uint32_t var7 = temp_loaded_i32 & 0xFFFFFF00;
        uint32_t value_to_store = var5 + var7;

        for (int b = 0; b < 4; b++) {
            memory[var1 + b] = (value_to_store >> (8 * b)) & 0xFF;
        }

        uint32_t call_param = var5 % 32;
        uint32_t func_index = var5 % 2;

        uint32_t val0 = 0, val4 = 0;
        for (int b = 0; b < 4; b++) val0 |= ((uint32_t)memory[b]) << (8 * b);
        for (int b = 0; b < 4; b++) val4 |= ((uint32_t)memory[4 + b]) << (8 * b);

        if (func_index == 0) { // rotate right
            val0 = (val0 >> call_param) | (val0 << (32 - call_param));
            val4 = (val4 >> call_param) | (val4 << (32 - call_param));
        } else { // rotate left
            val0 = (val0 << call_param) | (val0 >> (32 - call_param));
            val4 = (val4 << call_param) | (val4 >> (32 - call_param));
        }

        for (int b = 0; b < 4; b++) memory[b] = (val0 >> (8 * b)) & 0xFF;
        for (int b = 0; b < 4; b++) memory[4 + b] = (val4 >> (8 * b)) & 0xFF;
    }

    for (int i = 0; i < 8; i++) {
        if (memory[i] != memory[16 + i]) return 0;
    }
    return 1;
}

// Kernel brute-force
__global__ void brute_force_kernel(char *results, int max_results, const uint8_t *wasm_data_at_8, const uint8_t *wasm_data_at_16, uint64_t start_idx, uint64_t end_idx) {
    uint64_t idx = start_idx + (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= end_idx) return;

    char candidate[9] = {0};
    uint64_t tmp = idx;
    for (int i = 0; i < 8; i++) {
        candidate[i] = d_alphabet[tmp % alphabet_len];
        tmp /= alphabet_len;
    }
    candidate[8] = '\0';

    if (check_gpu(candidate, wasm_data_at_8, wasm_data_at_16)) {
        int pos = atomicAdd((int*)results, 1);
        if (pos < max_results - 1) {
            for (int i = 0; i < 9; i++) results[(pos + 1) * 9 + i] = candidate[i];
        }
    }
}

int main() {
    // Disable buffering
    setvbuf(stdout, NULL, _IOLBF, 0); // Output line by line

    uint8_t *d_wasm_data_at_8, *d_wasm_data_at_16;
    char *d_results;
    char *h_results;
    int max_results = 10;

    CUDA_CHECK(hipMalloc(&d_wasm_data_at_8, 8));
    CUDA_CHECK(hipMalloc(&d_wasm_data_at_16, 8));
    CUDA_CHECK(hipMemcpy(d_wasm_data_at_8, wasm_data_at_8, 8, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_wasm_data_at_16, wasm_data_at_16, 8, hipMemcpyHostToDevice));

    h_results = (char*)malloc((1 + max_results) * 9);
    CUDA_CHECK(hipMalloc(&d_results, (1 + max_results) * 9));
    CUDA_CHECK(hipMemset(d_results, 0, (1 + max_results) * 9));

    uint64_t total_combinations = 1;
    for (int i = 0; i < 8; i++) total_combinations *= alphabet_len;
    printf("[i] Total number of combinations : %llu combinaisons\n", total_combinations);

    int threads_per_block = 256;
    uint64_t chunk_size = (uint64_t)threads_per_block * 1024 * 1024; // ~256M par chunk

    printf("[i] Starting bruteforce...\n");
    clock_t start_bruteforce = clock();

    int found_any = 0;
    for (uint64_t start_idx = 0; start_idx < total_combinations; start_idx += chunk_size) {
        uint64_t end_idx = start_idx + chunk_size;
        if (end_idx > total_combinations) end_idx = total_combinations;

        int blocks_per_grid = (int)((end_idx - start_idx + threads_per_block - 1) / threads_per_block);

        brute_force_kernel<<<blocks_per_grid, threads_per_block>>>(d_results, max_results, d_wasm_data_at_8, d_wasm_data_at_16, start_idx, end_idx);

        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(h_results, d_results, (1 + max_results) * 9, hipMemcpyDeviceToHost));
        int found_count = *((int*)h_results);

        if (found_count > 0) {
            printf("\n[!][i] Password(s) found :\n");
            for (int i = 0; i < found_count; i++) {
                printf("  -> %s\n", &h_results[(i + 1) * 9]);
            }
            found_any = 1;
            break;
        }
    }
    
    if (!found_any) {
        printf("[X] No password found in the whole search space.\n");
    }

    clock_t end_bruteforce = clock();
    double elapsed_time = (double)(end_bruteforce - start_bruteforce) / CLOCKS_PER_SEC;
    printf("[i] Bruteforce completed in %.2f seconds.\n", elapsed_time);

    hipFree(d_wasm_data_at_8);
    hipFree(d_wasm_data_at_16);
    hipFree(d_results);
    free(h_results);

    return 0;
}
