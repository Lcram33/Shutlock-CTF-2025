#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

// Alphabet en mémoire constante (non utilisé ici, mais utile pour brute-force)
__constant__ char d_alphabet[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
const int alphabet_len = 62;

// WASM data (copiés sur GPU)
uint8_t wasm_data_at_8[] = {0xDC, 0x87, 0xDB, 0x6B, 0x7C, 0xFD, 0x6D, 0x20};
uint8_t wasm_data_at_16[] = {0x2C, 0xE4, 0x0E, 0x7B, 0x77, 0x02, 0x1A, 0x5D};

// Fonction GPU check corrigée
__device__ int check_gpu(const char *input_text, const uint8_t *wasm_data_at_8, const uint8_t *wasm_data_at_16) {
    uint8_t memory[24] = {0};

    // Initialisation mémoire
    for (int i = 0; i < 8; i++) memory[i] = input_text[i];
    for (int i = 0; i < 8; i++) memory[8 + i] = wasm_data_at_8[i];
    for (int i = 0; i < 8; i++) memory[16 + i] = wasm_data_at_16[i];

    for (int var1 = 0; var1 < 8; var1++) {
        uint8_t var5 = memory[var1] ^ memory[8 + var1];

        // Charger 4 octets à partir de var1 (i32.load)
        uint32_t temp_loaded_i32 = 0;
        for (int b = 0; b < 4; b++) {
            temp_loaded_i32 |= ((uint32_t)memory[var1 + b]) << (8 * b);
        }

        uint32_t var7 = temp_loaded_i32 & 0xFFFFFF00;
        uint32_t value_to_store = var5 + var7;

        // Stocker 4 octets à partir de var1 (i32.store)
        for (int b = 0; b < 4; b++) {
            memory[var1 + b] = (value_to_store >> (8 * b)) & 0xFF;
        }

        uint32_t call_param = var5 % 32;
        uint32_t func_index = var5 % 2;

        // Reconstituer les deux uint32_t pour rotation
        uint32_t val0 = 0, val4 = 0;
        for (int b = 0; b < 4; b++) val0 |= ((uint32_t)memory[b]) << (8 * b);
        for (int b = 0; b < 4; b++) val4 |= ((uint32_t)memory[4 + b]) << (8 * b);

        if (func_index == 0) {
            val0 = (val0 >> call_param) | (val0 << (32 - call_param));
            val4 = (val4 >> call_param) | (val4 << (32 - call_param));
        } else {
            val0 = (val0 << call_param) | (val0 >> (32 - call_param));
            val4 = (val4 << call_param) | (val4 >> (32 - call_param));
        }

        for (int b = 0; b < 4; b++) memory[b] = (val0 >> (8 * b)) & 0xFF;
        for (int b = 0; b < 4; b++) memory[4 + b] = (val4 >> (8 * b)) & 0xFF;
    }

    // Comparaison finale 64 bits
    for (int i = 0; i < 8; i++) {
        if (memory[i] != memory[16 + i]) return 0;
    }
    return 1;
}

// Kernel de test simple
__global__ void test_password_kernel(int *result, const uint8_t *wasm_data_at_8, const uint8_t *wasm_data_at_16) {
    const char test_pw[8] = {'p','a','s','s','w','o','r','d'};
    if (check_gpu(test_pw, wasm_data_at_8, wasm_data_at_16)) {
        *result = 1;
    } else {
        *result = 0;
    }
}

int main() {
    int *d_result;
    int h_result = 0;

    uint8_t *d_wasm_data_at_8, *d_wasm_data_at_16;

    hipMalloc(&d_result, sizeof(int));
    hipMalloc(&d_wasm_data_at_8, 8);
    hipMalloc(&d_wasm_data_at_16, 8);

    hipMemcpy(d_wasm_data_at_8, wasm_data_at_8, 8, hipMemcpyHostToDevice);
    hipMemcpy(d_wasm_data_at_16, wasm_data_at_16, 8, hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    test_password_kernel<<<1,1>>>(d_result, d_wasm_data_at_8, d_wasm_data_at_16);
    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    if (h_result == 1) {
        printf("Le mot de passe 'password' est reconnu comme valide.\n");
    } else {
        printf("Le mot de passe 'password' n'est PAS reconnu.\n");
    }

    hipFree(d_result);
    hipFree(d_wasm_data_at_8);
    hipFree(d_wasm_data_at_16);

    return 0;
}
